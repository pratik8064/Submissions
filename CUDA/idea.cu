#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define DEBUG

typedef unsigned long ulong;
typedef unsigned short ushort;

__device__ ushort add(long a, long b)
{
	return (ushort)((a + b) % 65536l);
}

__device__ ushort multiply(long a, long b)
{
	long ch, cl, c;

	if (a == 0) a = 65536l;
	if (b == 0) b = 65536l;
	c = a * b;
	if (c) {
		ch = (c >> 16) & 65535l;
		cl = c & 65535l;
		if (cl >= ch) return (ushort) (cl - ch);
		return (ushort) ((cl - ch + 65537l) & 65535l);
	}
	if (a == b) return 1;
	return 0;
}

__global__ void IDEA_encryption(ushort *X, ushort *Y, long *K)
{
	
	ushort a, r, t0, t1, t2;
	//ushort i;	
	int tid = threadIdx.x;
	for (r = 0; r < 8; r++) {
		X[tid * 4 + 0] = multiply(X[tid * 4 + 0], K[r*6+0]);
		X[tid * 4 + 3] = multiply(X[tid * 4 + 3], K[r*6+3]);
		X[tid * 4 + 1] = add(X[tid * 4 + 1], K[r*6+1]);
		X[tid * 4 + 2] = add(X[tid * 4 + 2], K[r*6+2]);
		t0 = multiply(K[r*6+4], X[tid * 4 + 0] ^ X[tid * 4 + 2]);
		t1 = multiply(K[r*6+5], add(t0, X[tid * 4 + 1] ^ X[tid * 4 + 3]));
		t2 = add(t0, t1);
		X[tid * 4 + 0] ^= t1;
		X[tid * 4 + 3] ^= t2;
		a = X[tid * 4 + 1] ^ t2;
		X[tid * 4 + 1] = X[tid * 4 + 2] ^ t1;
		X[tid * 4 + 2] = a;
		#ifdef DEBUG
		/*printf("%u ", r + 1);
		for (i = 0; i < 6; i++)
			printf("%hu ", (ushort) K[r*6+i]);
		printf("%hu %hu %hu %hu\n", X[0], X[1], X[2], X[3]);*/
		#endif
	}
	Y[tid * 4 + 0] = multiply(X[tid * 4 + 0], K[8*6+0]);
	Y[tid * 4 + 3] = multiply(X[tid * 4 + 3], K[8*6+3]);
	Y[tid * 4 + 1] = add(X[tid * 4 + 2], K[8*6+1]);
	Y[tid * 4 + 2] = add(X[tid * 4 + 1], K[8*6+2]);
	#ifdef DEBUG
	/*printf("9 ");
	for (i = 0; i < 6; i++)
		printf("%hu ", (ushort) K[8*6+i]);
	printf("%hu %hu %hu %hu\n", Y[0], Y[1], Y[2], Y[3]);*/
	#endif
}

__device__ ushort bits_to_ushort(ushort *bits)
{
	ushort i, value = bits[0];

	for (i = 1; i < 16; i++)
		value = (ushort) ((value << 1) + bits[i]);
	return value;
}

__device__ void ushort_to_bits(ushort number, ushort *bits)
{
	ushort i, temp[16];
	int tid = threadIdx.x;
	for (i = 0; i < 16; i++) {
		temp[i] = (ushort) (number & 1);
		number >>= 1;
	}
	/*for (i = 0; i < 16; i++)
		bits[i] = temp[15 - i];*/
	if(tid<16)
	bits[tid] = temp[15-tid];

}

__device__ void cyclic_left_shift(ushort index, ushort *bits1,
                       ushort *bits2, long *K)
{
	ushort i;
	int tid = threadIdx.x;
	if (index == 0) {
		for (i = 0; i < 6; i++)
			ushort_to_bits((ushort) K[0*6+i], bits1 + 16 * i);
		ushort_to_bits((ushort) K[1*6+0], bits1 + 96);
		ushort_to_bits((ushort) K[1*6+1], bits1 + 112);
	}
	/*i = 0;
	for (j = 25; j < 128; j++)
		bits2[i++] = bits1[j];
	for (j = 0; j < 25; j++)
		bits2[i++] = bits1[j];*/
	if( tid < 103)
  		bits2[tid] = bits1[tid + 25];
	else if(tid >= 103 && tid <128)
  		bits2[tid] = bits1[tid - 103];

	
	switch (index) {
		case 0 :
			/*for (i = 2; i < 6; i++)
				K[1][i] = bits_to_ushort(bits2 + 16 * (i - 2));
			for (i = 0; i < 4; i++)
				K[2][i] = bits_to_ushort(bits2 + 64 + 16 * i);*/
			if(tid >=2 && tid < 6)
				K[1*6+tid] = bits_to_ushort(bits2 + 16 * (tid - 2 ));
			if(tid < 4)
				K[2*6+tid] = bits_to_ushort(bits2 + 64 + 16 * tid);

		break;
		case 1 :
			K[2*6+4] = bits_to_ushort(bits2);
			K[2*6+5] = bits_to_ushort(bits2 + 16);
			/*for (i = 0; i < 6; i++)
				K[3][i] = bits_to_ushort(bits2 + 32 + 16 * i);*/
			if(tid < 6)
				K[3*6+tid] = bits_to_ushort(bits2 + 32 + 16 * tid);
		break;

		case 2 :
	
			/*for (i = 0; i < 6; i++)
				K[4*6+i] = bits_to_ushort(bits2 + 16 * i);*/
			if(tid < 6)
				K[4*6+tid] = bits_to_ushort(bits2 + 16 * tid);
			K[5*6+0] = bits_to_ushort(bits2 + 96);
			K[5*6+1] = bits_to_ushort(bits2 + 112);
		break;
		case 3 :
	
			/*for (i = 2; i < 6; i++)
				K[5][i] = bits_to_ushort(bits2 + 16 * (i - 2));
			for (i = 0; i < 4; i++)
				K[6][i] = bits_to_ushort(bits2 + 64 + 16 * i);*/
    
			if(tid >=2 && tid < 6)
				        K[5*6+tid] = bits_to_ushort(bits2 + 16 * (tid - 2));
			 if(tid < 4)
				        K[6*6+tid] = bits_to_ushort(bits2 + 64 + 16 * tid);
		break;
		case 4 :
			K[6*6+4] = bits_to_ushort(bits2);
			K[6*6+5] = bits_to_ushort(bits2 + 16);

			/*for (i = 0; i < 6; i++)
				K[7][i] = bits_to_ushort(bits2 + 32 + 16 * i);*/

			if(tid < 6)
				K[7*6+tid] = bits_to_ushort(bits2 + 32 + 16 * tid);
		break;
		case 5 :
			/*for (i = 0; i < 4; i++)
				K[8][i] = bits_to_ushort(bits2 + 16 * i);*/

			if(tid < 4)
		        K[8*6+tid] = bits_to_ushort(bits2 + 16 * tid);
		break;
	}
}

__global__ void IDEA_encryption_key_schedule(ushort *key, long *K)
{
	ushort bits1[128], bits2[128];
	int tid = threadIdx.x;

	//for (i = 0; i < 6; i++) K[0][i] = key[i];
	if(tid < 6)
	  	K[0*6+tid] = key[tid];
	if(tid == 32)
  		K[1*6+0] = key[6], K[1*6+1] = key[7];
  
	cyclic_left_shift(0, bits1, bits2, K);
	cyclic_left_shift(1, bits2, bits1, K);
	cyclic_left_shift(2, bits1, bits2, K);
	cyclic_left_shift(3, bits2, bits1, K);
	cyclic_left_shift(4, bits1, bits2, K);
	cyclic_left_shift(5, bits2, bits1, K);
	
}

__device__ void extended_euclidean(long a, long b, long *x, long *y, long *d)
{
	long q, r, x1, x2, y1, y2;

	if (b == 0) {
		*d = a, *x = 1, *y = 0;
		return;
	}
	x2 = 1, x1 = 0, y2 = 0, y1 = 1;
	while (b > 0) {
		q = a / b, r = a - q * b;
		*x = x2 - q * x1;
		*y = y2 - q * y1;
		a = b, b = r, x2 = x1, x1 = *x, y2 = y1, y1 = *y;
	}
	*d = a, *x = x2, *y = y2;
}

__device__ long inv(ushort ub)
{
	long d, a = 65537l, b = ub, x, y;

	if (ub == 0) return 65536l;
	extended_euclidean(a, b, &x, &y, &d);
	if (y >= 0) return (ushort) y;
	return (ushort) (y + 65537l);
}

__global__ void IDEA_decryption_key_schedule(long *K, long *L)
{
	ushort r8, r9;
	int tid = threadIdx.x;
	if(tid == 0){
		L[0*6+0] = inv((ushort) K[8*6+0]);
		L[0*6+1] = - K[8*6+1];
		L[0*6+2] = - K[8*6+2];
		L[0*6+3] = inv((ushort) K[8*6+3]);
		L[0*6+4] =  K[7*6+4];
		L[0*6+5] =  K[7*6+5];
}

	if(tid>0 && tid <8){
    		r9 = (ushort) (8 - tid);
    		r8 = (ushort) (7 - tid);
    		L[tid*6+0] = inv((ushort) K[r9*6+0]);
    		L[tid*6+1] = - K[r9*6+2];
    		L[tid*6+2] = - K[r9*6+1];
    		L[tid*6+3] = inv((ushort) K[r9*6+3]);
    		L[tid*6+4] = K[r8*6+4];
    		L[tid*6+5] = K[r8*6+5];
	}

	if(tid == 0){
		L[8*6+0] = inv((ushort) K[0*6+0]);
		L[8*6+1] = - K[0*6+1];
		L[8*6+2] = - K[0*6+2];
		L[8*6+3] = inv((ushort) K[0*6+3]);
		L[8*6+4] = L[8*6+6] = 0;
	}
}

int main(int argc, char *argv[])
{	
	/*int devcount;
	cudaGetDeviceCount(&devcount);
	printf("%d ",devcount);
	for(int i=0;i<devcount;i++){
		cudaDeviceProp prop;
		cudaGetDeviceProperties(&prop , i);
		printf("%d ",prop.warpSize);
	}*/
	int tc=512;
	int count;
	long *K, *L;K=NULL;L=NULL;
	long *dev_K , *dev_L; dev_K=NULL; dev_L = NULL;
	ushort key[8] = {1, 2, 3, 4, 5, 6, 7, 8};
	ushort *X, *Y;
	ushort *dev_X, *dev_Y;
	ushort *dev_key = NULL;
  
	FILE *in_file  = fopen("/home/jignesh/input.jpg", "r"); 
	FILE *out_file = fopen("/home/jignesh/output.jpg", "w"); 
	FILE *mid_w = fopen("/home/jignesh/encrypt.jpg", "w");
	if (in_file == NULL || out_file == NULL) 
	{
		printf("Error! Could not open file\n"); 
		exit(-1); 
	}
	
	K = (long *)malloc(54 * sizeof(long ));
	L = (long *)malloc(54 * sizeof(long ));

	hipMalloc((void **) &dev_key , 16);
	hipMemcpy( dev_key, key, 16, hipMemcpyHostToDevice);

 	size_t pitch;
	hipMallocPitch(&dev_K, &pitch, sizeof(long)*9, 6);
	hipMallocPitch(&dev_L, &pitch, sizeof(long)*9, 6);
	
	IDEA_encryption_key_schedule<<<1,128>>>(dev_key, dev_K);

	hipMemcpy( K , dev_K, 9*6* sizeof(long *),hipMemcpyDeviceToHost);

	X = (ushort *)malloc(tc * 4 * sizeof(ushort));
	Y = (ushort *)malloc(tc * 4 * sizeof(ushort));

	count = fread( X,2,tc*4,in_file);

	hipMalloc((void **) &dev_Y , tc*4*2);
	hipMalloc((void **) &dev_X , tc*4*2);	

	while(count==tc * 4){
	   	hipMemcpy( dev_X, X,tc * 8, hipMemcpyHostToDevice);
   
		IDEA_encryption<<<1,tc>>>(dev_X, dev_Y, dev_K);

		hipMemcpy( Y, dev_Y,tc * 8, hipMemcpyDeviceToHost);

		fwrite( Y,2,tc*4,mid_w);
		count = fread( X,2,tc*4,in_file);
	}
	if(count < tc*4 && count > 0){
		fwrite( X,2,count,mid_w);
	}
	
	fclose(mid_w);
	FILE *mid_file = fopen("/home/jignesh/encrypt.jpg", "r");

	IDEA_decryption_key_schedule<<<1,8>>>(dev_K, dev_L);

	count = fread( Y,2,tc*4,mid_file);
	while(count==tc * 4){
		hipMemcpy( dev_Y, Y,tc * 8, hipMemcpyHostToDevice);
	
		IDEA_encryption<<<1,tc>>>(dev_Y, dev_X, dev_L);
	
		hipMemcpy( X, dev_X,tc * 8, hipMemcpyDeviceToHost);
 
		fwrite( X,2,tc*4,out_file);
		count = fread( Y,2,tc*4,mid_file);
	}
	if(count < tc*4 && count > 0){
		fwrite( Y,2,count,out_file);
	} 
	free(K);
	free(L);
	return 0;
}
